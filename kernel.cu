
#include <hip/hip_runtime.h>
__global__ void createVertices_kernel(float4* positions, float time,
  unsigned int width, unsigned int height)
{
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  // Calculate uv coordinates
  float u = x / (float)width;
  float v = y / (float)height;
  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;
  // calculate simple sine wave pattern
  float freq = 4.0f;
  float w = sinf(u * freq + time)
  * cosf(v * freq + time) * 0.5f;
  // Write positions
  positions[y * width + x] = make_float4(u, w, v, 1.0f);
}

void createVertices(void * positions, float time,
  unsigned int width, unsigned int height)
{
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
  createVertices_kernel<<<dimGrid, dimBlock>>>((float4*)positions, time,
    width, height);
  hipDeviceSynchronize();
}
